#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void reduceToDiagonal(double *mat, int n, int i) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();
    if (j < n) {
        if (j != i) {
            double d = mat[j * 2*n + i] / mat[i * 2*n + i];
            for (int k = 0; k < 2*n; ++k) {
                mat[j * 2*n + k] -= mat[i * 2*n + k] * d;
            }
            __syncthreads();        
        }
    }
}

__global__ void reduceToUnit(double *mat, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __syncthreads();
    if (i < n) {
        double d = mat[i * 2 * n + i];
        for (int j = 0; j < 2 * n; ++j) {
            mat[i * 2*n + j] = mat[i * 2*n + j] / d;
        }
        __syncthreads();
    }
}

int main() {
    int n;
    double *mat_host, *mat_device;
    int mat_size = 2 * n * n * sizeof(double);

    cin >> n;

    // Allocating memory for matrix host
    mat_host = new double[2 * n * n];

    // Inputs the coefficients of the matrix
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            cin >> mat_host[i * 2*n + j];
        }
    }

    // Initializing Right-hand side to identity matrix
    for (int i = 0; i < n; ++i) {
        for (int j = n; j < 2*n; ++j) {
            if (j == (i + n)) {
                mat_host[i * 2*n + j] = 1;
            } else {
                mat_host[i * 2*n + j] = 0;
            }
        }
    }

    // Allocating memory for matrix device
    hipMalloc((void **)&mat_device, mat_size);

    hipMemcpy(mat_device, mat_host, mat_size, hipMemcpyHostToDevice);

    dim3 blockShape = dim3(32);
    dim3 gridShape = dim3(max(1.0, ceil((double) n / (double)blockShape.x)));

    // Reducing To Diagonal Matrix
    for(int i = 0; i < n; ++i)
    {
        reduceToDiagonal<<<gridShape, blockShape>>>(mat_device, n, i);
    }
    
    // Reducing To Unit Matrix
    reduceToUnit<<<gridShape, blockShape>>>(mat_device, n) ;

    hipMemcpy(mat_host, mat_device, mat_size, hipMemcpyDeviceToHost);

    cout << n << endl;
    for (int i = 0; i < n; ++i) {
        for (int j = n; j < 2 * n; ++j) {
            cout << mat_host[i * 2 * n + j] << " ";
        }
        cout << endl;
    }

    // Deleting the memory allocated
    delete[] mat_host;
    hipFree(mat_device);

    return 0;
}
