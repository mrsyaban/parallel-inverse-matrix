#include <iostream>
#include <hip/hip_runtime.h>


__global__ void reduceToDiagonal(double *mat, int n) {
}

__global__ void reduceToUnit(double *mat, int n) {
}

int main() {
    int n;
    std::cin >> n;

    double *matHost, *matDevice;
    int size = 2 * n * n * sizeof(double);

    matHost = new double[2 * n * n];
    hipMalloc((void **)&matDevice, size);

    // Inputs the coefficients of the matrix
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cin >> matHost[i * n + j];
        }
    }

    // Initializing Right-hand side to identity matrix
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < 2 * n; ++j) {
            if (j == (i + n)) {
                matHost[i * 2 * n + j] = 1;
            } else {
                matHost[i * 2 * n + j] = 0;
            }
        }
    }

    hipMemcpy(matDevice, matHost, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Reducing To Diagonal Matrix kernel
    reduceToDiagonal<<<blocksPerGrid, threadsPerBlock>>>(matDevice, n);

    // Reducing To Unit Matrix kernel
    reduceToUnit<<<blocksPerGrid, threadsPerBlock>>>(matDevice, n);

    hipMemcpy(matHost, matDevice, size, hipMemcpyDeviceToHost);

    std::cout << n << std::endl;
    for (int i = 0; i < n; ++i) {
        for (int j = n; j < 2 * n; ++j) {
            std::cout << matHost[i * 2 * n + j] << " ";
        }
        std::cout << std::endl;
    }

    delete[] matHost;
    hipFree(matDevice);

    return 0;
}
